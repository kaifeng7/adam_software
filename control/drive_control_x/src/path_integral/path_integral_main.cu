#include "hip/hip_runtime.h"
#define __CUDACC_VER__ __CUDACC_VER_MAJOR__ * 10000 + __CUDACC_VER_MINOR__ * 100 + __CUDACC_VER_BUILD__

#include "drive_control_x/path_integral/meta_math.h"
#include "drive_control_x/path_integral/param_getter.h"
#include "drive_control_x/path_integral/adam_plant.h"
#include "drive_control_x/PathIntegralParamsConfig.h"
#include "drive_control_x/path_integral/costs.cuh"

//Including neural net model
#ifdef MPPI_NNET_USING_CONSTANT_MEM__
__device__ __constant__ float NNET_PARAMS[param_counter(6,32,32,4)];
#endif

#include "drive_control_x/path_integral/neural_net_model.cuh"
#include "drive_control_x/path_integral/car_bfs.cuh"
#include "drive_control_x/path_integral/car_kinematics.cuh"
#include "drive_control_x/path_integral/generalized_linear.cuh"
#include "drive_control_x/path_integral/mppi_controller.cuh"
#include "drive_control_x/path_integral/run_control_loop.cuh"

#include <ros/ros.h>
#include <atomic>
#include <math.h> 

#include <stdlib.h>
#include <stdio.h>

using namespace drive_control;

#ifdef USE_NEURAL_NETWORK_MODEL__ /*Use neural network dynamics model*/
const int MPPI_NUM_ROLLOUTS__ = 1920;
const int BLOCKSIZE_X = 8;
const int BLOCKSIZE_Y = 16;
typedef NeuralNetModel<7,2,3,6,32,32,4> DynamicsModel;
#elif USE_BASIS_FUNC_MODEL__ /*Use the basis function model* */
const int MPPI_NUM_ROLLOUTS__ = 2560;
const int BLOCKSIZE_X = 16;
const int BLOCKSIZE_Y = 4;
typedef GeneralizedLinear<CarBasisFuncs, 7, 2, 25, CarKinematics, 3> DynamicsModel;
#endif

//Convenience typedef for the MPPI Controller.
typedef MPPIController<DynamicsModel, MPPICosts, MPPI_NUM_ROLLOUTS__, BLOCKSIZE_X, BLOCKSIZE_Y> Controller;

int main(int argc, char** argv){
    //Ros node initialization
    ros::init(argc, argv, "mppi_controller");
    
    ros::NodeHandle mppi_node("~");

    //Load setup parameters
    SystemParams params;
    loadParams(&params, mppi_node);

    //Define the mppi costs
    MPPICosts* costs = new MPPICosts(mppi_node);

    //Define the internal dynamics model for mppi
    float2 control_constraints[2] = {make_float2(-.99, .99), make_float2(-.99, params.max_throttle)};
    DynamicsModel* model = new DynamicsModel(1.0/params.hz, control_constraints);
    model->loadParams(params.model_path); //Load the model parameters from the launch file specified path//Define the internal dynamics model for mppi


    int optimization_stride = getRosParam<int>("optimization_stride", mppi_node);

    //Define the controller
    float init_u[2] = {(float)params.init_steering, (float)params.init_throttle};
    float exploration_std[2] = {(float)params.steering_std, (float)params.throttle_std};
    Controller* mppi = new Controller(model, costs, params.num_timesteps, params.hz, params.gamma, exploration_std, 
                                        init_u, params.num_iters, optimization_stride);

    AdamPlant* robot = new AdamPlant(mppi_node, mppi_node, params.debug_mode, params.hz, false);

    //Setup dynamic reconfigure callback
    dynamic_reconfigure::Server<PathIntegralParamsConfig> server;
    dynamic_reconfigure::Server<PathIntegralParamsConfig>::CallbackType callback_f;
    callback_f = boost::bind(&AdamPlant::dynRcfgCall, robot, _1, _2);
    server.setCallback(callback_f);

    boost::thread optimizer;

    std::atomic<bool> is_alive(true);
    optimizer = boost::thread(&runControlLoop<Controller>, mppi, robot, &params, &mppi_node, &is_alive);

    ros::spin();

    //Shutdown procedure
    is_alive.store(false);
    optimizer.join();
    robot->shutdown();
    mppi->deallocateCudaMem();
    delete robot;
    delete mppi;
    delete costs;
    delete model;
}